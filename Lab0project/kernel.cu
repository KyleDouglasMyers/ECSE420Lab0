#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "lodepng.h"
#include <stdio.h>



__global__ void rectifyKernel(unsigned char* managed_image, int batch)
{
	//int threadId = threadIdx.x + blockIdx.x * blockDim.x;

	printf("Thread %d\n", threadIdx.x);

	int x = threadIdx.x * batch;

	for (int i = x; i < x + batch; i++) {
		if (managed_image[i] < 127) {
			managed_image[i] = 127;
		}
	}
}


void rectify(char* input_filename, char* output_filename, int threads)
{
	unsigned error;
	unsigned char* image, * new_image, * managed_image, * managed_new_image;
	unsigned width, height;

	error = lodepng_decode32_file(&image, &width, &height, input_filename);
	if (error) printf("error %u: %s\n", error, lodepng_error_text(error));

	//malloc some space in gpu memory

	int batch = width * height * 4 / threads;

	hipMalloc((void**)&managed_image, width * height * 4 * sizeof(unsigned char));
	
	hipMalloc((void**)&managed_new_image, width * height * 4 * sizeof(unsigned char)); 
	
	hipMemcpy(managed_image, image, width * height * 4 * sizeof(unsigned char), hipMemcpyHostToDevice);



	//CUDA call
	rectifyKernel <<< 1, threads >>> (managed_image, batch);

	hipMemcpy(image, managed_image, width * height * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);

	lodepng_encode32_file(output_filename, image, width, height);

	free(image);
	//free(new_image);
}


int main()
{
	char input_filename[] = "Test_1.png";
	char rectify_output_filename[] = "myRectifiedPic.png";
	char pool_output_filename[] = "myPooledPic.png";


	rectify(input_filename, rectify_output_filename, 4);

	return 0;
}